#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/stat.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <argparse/argparse.h>
#include <grid/grid_3d.h>
#include <topography/topography.h>
#include <topography/initializations/constant.h>
#include <topography/initializations/random.h>
#include <topography/initializations/linear.h>
#include <topography/initializations/quadratic.h>
#include <topography/initializations/cerjan.h>
#include <test/check.h>
#include <test/grid_check.h>
#include <mpi/partition.h>
#include <vtk/vtk.h>

#ifdef USE_OPTIMIZED_KERNELS
#include <topography/velocity.cuh>
#include <topography/stress_attenuation.cuh>
#else
#include <topography/topography.cuh>
#endif
#include <topography/geometry.h>
#include <topography/host.h>
 
static const char *const usages[] = {
    "topography_kernels [options] [[--] args]",
    "topography_kernels [options]",
    NULL,
};

static topo_t reference;
static int px = 0;
static int py = 0;
static int nx = 0;
static int ny = 0;
static int nz = 0;
static int nt = 0;
static prec h = 1.0;
static prec dt = 0.25;
static int coord[2] = {0, 0};
static int dim[2] = {0, 0};
static int rank, size;
static struct side_t side;
static hipStream_t stream_1, stream_2, stream_i;
static int use_optimized_kernels = USE_OPTIMIZED_KERNELS;
static const char *outputdir;
static const char *inputdir;
static int run_velocity = 1;
static int run_stress = 1;

void init(topo_t *T);
void run(topo_t *T);
void write(topo_t *host, const char *outputdir);
void write_file(const char *path, const char *filename, const _prec *data, const
                int size);

int compare(topo_t *host, const char *inputdir);
void read_file(const char *path, const char *filename, _prec *data, const int
                size);

int main(int argc, char **argv)
{
        MPI_Init(&argc, &argv);
        MPI_Comm_set_errhandler(MPI_COMM_WORLD, MPI_ERRORS_RETURN); 
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &size);

        printf("Optimized kernels: %d.\n", use_optimized_kernels);
        hipStreamCreate(&stream_1);
        hipStreamCreate(&stream_2);
        hipStreamCreate(&stream_i);

        struct argparse_option options[] = {
            OPT_HELP(),
            OPT_GROUP("Options"),
            OPT_INTEGER('p', "px", &px,
                        "Number of processes in the X-direction", NULL, 0, 0),
            OPT_INTEGER('q', "py", &py,
                        "Number of processes in the Y-direction", NULL, 0, 0),
            OPT_INTEGER('x', "nx", &nx,
                        "Number of grid points in the X-direction", NULL, 0, 0),
            OPT_INTEGER('y', "ny", &ny,
                        "Number of grid points in the Y-direction", NULL, 0, 0),
            OPT_INTEGER('z', "nz", &nz,
                        "Number of grid points in the Z-direction", NULL, 0, 0),
            OPT_INTEGER('t', "nt", &nt,
                        "Number of iterations to perform", NULL, 0, 0),
            OPT_STRING('o', "output", &outputdir,
                        "Write results to output directory", NULL, 0, 0),
            OPT_STRING('i', "input", &inputdir,
                        "Read results from input directory", NULL, 0, 0),
            OPT_INTEGER('s', "stress", &run_stress,
                        "Run stress kernels", NULL, 0, 0),
            OPT_INTEGER('v', "velocity", &run_velocity,
                        "Run velocity kernels", NULL, 0, 0),
            OPT_END(),
        };

        struct argparse argparse;
        argparse_init(&argparse, options, usages, 0);
        argparse_describe(
            &argparse,
            "\nPerformance analysis of CUDA compute kernels for AWP.", "\n");
        argc = argparse_parse(&argparse, argc, (const char**)argv);

        dim[0] = px;
        dim[1] = py;
        
        int period[2] = {0, 0};
        int err = 0;
        MPI_Comm comm;
        err = mpi_partition_2d(rank, dim, period, coord, &side, &comm);
        assert(err == 0);

        topo_t device;
        topo_t host;

        init(&device);
        host = device;
        topo_h_malloc(&host);
        hipProfilerStart();
        run(&device);
        hipDeviceSynchronize();
        hipProfilerStop();

        topo_dtoh(&host, &device);
        write(&host, outputdir);
        err = compare(&host, inputdir);

        topo_h_free(&host);
        topo_d_free(&device);
        topo_free(&device);

        MPI_Finalize();

        return 0;
}

void init(topo_t *T)
{
        *T = topo_init(1, "", rank, side.left, side.right, side.front,
                              side.back, coord, px, py, nx, ny, nz, dt, h,
                              stream_1, stream_2, stream_i);
        topo_d_malloc(T);
        topo_d_zero_init(T);
        topo_d_cerjan_disable(T);
        topo_init_metrics(T);
        topo_init_grid(T);

        // Gaussian hill geometry
        _prec3_t hill_width = {.x = (_prec)nx / 2, .y = (_prec)ny / 2, .z = 0};
        _prec hill_height = 0.1;
        _prec3_t hill_center = {.x = 0, .y = 0, .z = 0};
        // No canyon
        _prec3_t canyon_width = {.x = 100, .y = 100, .z = 0};
        _prec canyon_height = 0;
        _prec3_t canyon_center = {.x = 0, .y = 0, .z = 0};
        topo_init_gaussian_hill_and_canyon(T, hill_width, hill_height,
                                           hill_center, canyon_width,
                                           canyon_height, canyon_center);

        // Set random initial conditions using fixed seed
        
        topo_d_random(T, 0, T->u1);
        topo_d_constant(T, 0, T->v1);
        topo_d_constant(T, 0, T->w1);

        topo_d_constant(T, 0, T->xx);
        topo_d_constant(T, 0, T->yy);
        topo_d_constant(T, 0, T->zz);
        topo_d_constant(T, 0, T->xy);
        topo_d_constant(T, 0, T->xz);
        topo_d_constant(T, 0, T->yz);

        topo_d_constant(T, 0, T->r1);
        topo_d_constant(T, 0, T->r2);
        topo_d_constant(T, 0, T->r3);
        topo_d_constant(T, 0, T->r4);
        topo_d_constant(T, 0, T->r5);
        topo_d_constant(T, 0, T->r6);

        topo_d_constant(T, 0, T->qpi);
        topo_d_constant(T, 0, T->qsi);
        
        topo_d_constant(T, 1.0, T->dcrjx);
        topo_d_constant(T, 1.0, T->dcrjy);
        topo_d_constant(T, 1.0, T->dcrjz);

        topo_d_constant(T, 0.5, T->wwo);
        topo_d_constanti(T, 1, T->ww);
        topo_d_constant(T, 0.4, T->vx1);
        topo_d_constant(T, 0.4, T->vx2);
        topo_d_constant(T, 0.4, T->coeff);

        topo_d_constant(T, 1, T->mui);
        topo_d_constant(T, 1, T->lami);
        topo_d_constant(T, 5, T->lam_mu);

        topo_build(T);

#if USE_OPTIMIZED_KERNELS
        printf("Setting constants\n");
        topo_set_constants(T);
#endif
}

void run(topo_t *T)
{
        for(int iter = 0; iter < nt; ++iter) {
                if (run_velocity) {
                        topo_velocity_interior_H(T);
                        //topo_velocity_front_H(T);
                        //topo_velocity_back_H(T);
                }

                CUCHK(hipStreamSynchronize(T->stream_1));
                CUCHK(hipStreamSynchronize(T->stream_2));
                CUCHK(hipStreamSynchronize(T->stream_i));
                hipDeviceSynchronize();

                if (run_stress) {
                        topo_stress_interior_H(T);
                        //topo_stress_left_H(T);
                        //topo_stress_right_H(T);
                }

                CUCHK(hipStreamSynchronize(T->stream_1));
                CUCHK(hipStreamSynchronize(T->stream_2));
                CUCHK(hipStreamSynchronize(T->stream_i));
                hipDeviceSynchronize();
        }
}

void write(topo_t *host, const char *outputdir)
{
        if (!outputdir) {
                return;
        }
        printf("writing to directory: %s \n", outputdir);

        mkdir(outputdir, 0700);

        int size = host->mx * host->my * host->mz;
        write_file(outputdir, "vx.bin", host->u1, size);
        write_file(outputdir, "vy.bin", host->v1, size);
        write_file(outputdir, "vz.bin", host->w1, size);
        write_file(outputdir, "xx.bin", host->xx, size);
        write_file(outputdir, "yy.bin", host->yy, size);
        write_file(outputdir, "zz.bin", host->zz, size);
        write_file(outputdir, "xy.bin", host->xy, size);
        write_file(outputdir, "xz.bin", host->xz, size);
        write_file(outputdir, "yz.bin", host->yz, size);
}

void write_file(const char *path, const char *filename, const _prec *data,
                const int size) 
{
        char output[512];
        sprintf(output, "%s/%s", path, filename);
        FILE *fh = fopen(output, "wb");
        if (!fh) {
                printf("Unable to open: %s. \n", filename);
                MPI_Abort(MPI_COMM_WORLD, 1);
                exit(1);
        }
        fwrite(data, sizeof(prec), size, fh);
        fclose(fh);

}

int compare(topo_t *host, const char *inputdir)
{
        if (!inputdir) {
                return 0;
        }

        topo_t reference = *host;
        topo_h_malloc(&reference);

        int size = host->mx * host->my * host->mz;
        printf("reading from directory: %s \n", inputdir);
        read_file(inputdir, "vx.bin", reference.u1, size);
        read_file(inputdir, "vy.bin", reference.v1, size);
        read_file(inputdir, "vz.bin", reference.w1, size);
        read_file(inputdir, "xx.bin", reference.xx, size);
        read_file(inputdir, "yy.bin", reference.yy, size);
        read_file(inputdir, "zz.bin", reference.zz, size);
        read_file(inputdir, "xy.bin", reference.xy, size);
        read_file(inputdir, "xz.bin", reference.xz, size);
        read_file(inputdir, "yz.bin", reference.yz, size);

        prec *a[9] = {reference.u1, reference.v1, reference.w1, reference.xx,
                reference.yy, reference.zz, reference.xy, reference.xz,
                reference.yz};
        prec *b[9] = {host->u1, host->v1, host->w1, host->xx, host->yy,
                host->zz, host->xy, host->xz, host->yz};
        const char *names[9] = {"vx", "vy", "vz", "xx", "yy", "zz", "xy", "xz",
        "yz"};
        double err[9];
        double total_error = 0;
        int nxt = nx - ngsl;
        int nyt = ny - ngsl;
        int nzt = nz;
        int excl = 0;
        int nbnd = 0;
        int i0 = 0;
        int in = i0 + nxt;
        int j0 = 0;
        int jn = j0 + nyt;
        int k0 = align;
        int kn = k0 + nzt;
        int new_size = (in - i0) * (jn - j0) * (kn - k0);
        total_error = 0;
        printf("slice: %d line: %d, %d %d \n", host->slice, host->line,
                        (2 * align + nz) * (4 + 2 * ngsl + ny), (2 * align +
                                nz));
        printf("Comparing in region [%d %d %d] [%d %d %d], size = %d \n", i0, j0, k0,
                        in, jn, kn,  new_size);
        for (int i = 0; i < 9; ++i) {
             printf("%s: \n", names[i]);
             err[i] = check_flinferr(a[i], b[i], 
                             i0, in, j0, jn, k0, kn,
                             host->line, host->slice);
                total_error += err[i];
        }

        for (int i = 0; i < 9; ++i) {
                printf("%s: %g ", names[i], err[i]);
        }
        printf("\n");


        int3_t grid_size = {nx, ny, nz};
        int3_t shift = {0, 0, 0};
        int3_t coordinate = {0,0,0};
        int3_t bnd1 = {0,0,0};
        int3_t bnd2 = {0,0,0};
        int padding = ngsl;
        prec gridspacing = 1.0;

        grid3_t grid = grid_init(grid_size, shift, coordinate, bnd1, bnd2, padding,
                        gridspacing);

        float *x1 = (float*)malloc(sizeof(x1) * grid.mem.x);
        float *y1 = (float*)malloc(sizeof(y1) * grid.mem.y);
        float *z1 = (float*)malloc(sizeof(z1) * grid.mem.z);
        grid_fill_x(x1, grid);
        grid_fill_y(y1, grid);
        grid_fill_z(z1, grid);
        int mem = grid.mem.x * grid.mem.y * grid.mem.z;
        float *x = (float*)malloc(sizeof(x) * mem);
        float *y = (float*)malloc(sizeof(y) * mem);
        float *z = (float*)malloc(sizeof(z) * mem);
        grid_fill3_x(x, x1, grid);
        grid_fill3_y(y, y1, grid);
        grid_fill3_z(z, z1, grid);
        vtk_write_grid("awp.vtk", x, y, z, grid);
        vtk_append_scalar("awp.vtk", "xx", host->xx, grid);
        
        vtk_write_grid("reference.vtk", x, y, z, grid);
        vtk_append_scalar("reference.vtk", "xx", reference.xx, grid);

        topo_h_free(&reference);
        return total_error > 1e-6;
}

void read_file(const char *path, const char *filename, _prec *data, const int
                size)
{
        char input[512];
        sprintf(input, "%s/%s", path, filename);

        FILE *fh = fopen(input, "rb");
        if (!fh) {
                printf("Unable to open: %s. \n", filename);
                MPI_Abort(MPI_COMM_WORLD, 1);
                exit(1);
        }
        int count = fread(data, sizeof(prec), size, fh);
        fclose(fh);
}
