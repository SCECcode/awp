#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/stat.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <argparse/argparse.h>
#include <topography/topography.h>
#include <topography/initializations/constant.h>
#include <topography/initializations/linear.h>
#include <topography/initializations/random.h>
#include <topography/initializations/cerjan.h>
#include <test/check.h>
#include <test/grid_check.h>
#include <mpi/partition.h>

#include <awp/kernel.h>

#include <topography/velocity.cuh>
#include <topography/stress_attenuation.cuh>
#include <topography/geometry.h>
#include <topography/host.h>
 
static const char *const usages[] = {
    "topography_kernels [options] [[--] args]",
    "topography_kernels [options]",
    NULL,
};

static topo_t reference;
static int px = 0;
static int py = 0;
static int nx = 64;
static int ny = 64;
static int nz = 64;
static int nt = 10;
static prec h = 1.0;
static prec dt = 0.5/3;
static int coord[2] = {0, 0};
static int dim[2] = {0, 0};
static int rank, size;
static struct side_t side;
static hipStream_t stream_1, stream_2, stream_i;

void init(topo_t *T);
void init_awp(topo_t *T);
void run(topo_t *T, topo_t *awp);
void write(topo_t *host, const char *outputdir);
void write_file(const char *path, const char *filename, const _prec *data, const
                int size);

int compare(topo_t *host, topo_t *awp);
void read_file(const char *path, const char *filename, _prec *data, const int
                size);

int main(int argc, char **argv)
{
        MPI_Init(&argc, &argv);
        MPI_Comm_set_errhandler(MPI_COMM_WORLD, MPI_ERRORS_RETURN); 
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &size);

        hipStreamCreate(&stream_1);
        hipStreamCreate(&stream_2);
        hipStreamCreate(&stream_i);

        struct argparse_option options[] = {
            OPT_HELP(),
            OPT_GROUP("Options"),
            OPT_INTEGER('p', "px", &px,
                        "Number of processes in the X-direction", NULL, 0, 0),
            OPT_INTEGER('q', "py", &py,
                        "Number of processes in the Y-direction", NULL, 0, 0),
            OPT_INTEGER('x', "nx", &nx,
                        "Number of grid points in the X-direction", NULL, 0, 0),
            OPT_INTEGER('y', "ny", &ny,
                        "Number of grid points in the Y-direction", NULL, 0, 0),
            OPT_INTEGER('z', "nz", &nz,
                        "Number of grid points in the Z-direction", NULL, 0, 0),
            OPT_INTEGER('t', "nt", &nt,
                        "Number of iterations to perform", NULL, 0, 0),
            OPT_END(),
        };

        struct argparse argparse;
        argparse_init(&argparse, options, usages, 0);
        argparse_describe(
            &argparse,
            "\nTest of Frequency dependent Q for AWP-TOPO.", "\n");
        argc = argparse_parse(&argparse, argc, (const char**)argv);

        dim[0] = px;
        dim[1] = py;
        
        int period[2] = {0, 0};
        int err = 0;
        MPI_Comm comm;
        err = mpi_partition_2d(rank, dim, period, coord, &side, &comm);
        assert(err == 0);

        topo_t topo, topo_h;
        topo_t awp, awp_h;

        init(&topo);
        init(&awp);
        init_awp(&awp);
        run(&topo, &awp);
        hipDeviceSynchronize();

        topo_h = topo;
        awp_h = awp;
        topo_h_malloc(&topo_h);
        topo_h_malloc(&awp_h);

        topo_dtoh(&topo_h, &topo);
        topo_dtoh(&awp_h, &awp);

        err = compare(&topo_h, &awp_h);

        topo_d_free(&topo);
        topo_d_free(&awp);
        topo_h_free(&topo_h);
        topo_h_free(&awp_h);
        topo_free(&topo);
        topo_free(&awp);

        MPI_Finalize();

        return err;
}

void init(topo_t *T)
{
        *T = topo_init(1, "", rank, side.left, side.right, side.front,
                              side.back, coord, px, py, nx, ny, nz, dt, h,
                              stream_1, stream_2, stream_i);
        topo_d_malloc(T);
        topo_d_zero_init(T);
        topo_d_cerjan_disable(T);
        topo_init_metrics(T);
        topo_init_grid(T);

        // Gaussian hill geometry
        _prec3_t hill_width = {.x = (_prec)nx / 2, .y = (_prec)ny / 2, .z = 0};
        _prec hill_height = 0;
        _prec3_t hill_center = {.x = 0, .y = 0, .z = 0};
        // No canyon
        _prec3_t canyon_width = {.x = 100, .y = 100, .z = 0};
        _prec canyon_height = 0;
        _prec3_t canyon_center = {.x = 0, .y = 0, .z = 0};
        topo_init_gaussian_hill_and_canyon(T, hill_width, hill_height,
                                           hill_center, canyon_width,
                                           canyon_height, canyon_center);

        // Set random initial conditions using fixed seed
        topo_d_linear_i(T, T->u1);
        topo_d_constant(T, 0, T->v1);
        topo_d_constant(T, 0, T->w1);

        topo_d_constant(T, 1, T->xx);
        topo_d_random(T, 5, T->yy);
        topo_d_random(T, 6, T->zz);
        topo_d_random(T, 7, T->xy);
        topo_d_random(T, 8, T->xz);
        topo_d_random(T, 9, T->yz);

        topo_d_constant(T, 1, T->r1);
        topo_d_constant(T, 0, T->r2);
        topo_d_constant(T, 0, T->r3);
        topo_d_constant(T, 0, T->r4);
        topo_d_constant(T, 0, T->r5);
        topo_d_constant(T, 0, T->r6);

        topo_d_constant(T, 0, T->qpi);
        topo_d_constant(T, 0, T->qsi);
        
        topo_d_constant(T, 1.0, T->dcrjx);
        topo_d_constant(T, 1.0, T->dcrjy);
        topo_d_constant(T, 1.0, T->dcrjz);

        topo_d_constant(T, 1.0, T->wwo);
        topo_d_constanti(T, 1, T->ww);
        topo_d_constant(T, 1.0, T->vx1);
        topo_d_constant(T, 1.0, T->vx2);
        topo_d_constant(T, 1.0, T->coeff);

        topo_d_random(T, 1, T->mui);
        topo_d_random(T, 1, T->lami);
        topo_d_constant(T, 1, T->mui);
        topo_d_constant(T, 1, T->lami);
        topo_d_constant(T, 1, T->lam_mu);
        topo_build(T);

        topo_set_constants(T);
}

void init_awp(topo_t *T)
{
        _prec fmajor = 0, fminor = 0, Rz[9], RzT[9];
        printf("Initializing AWP: %d %d %d\n", nx, ny, nz);
        SetDeviceConstValue(&h, dt, &nx, &ny, &nz, 1, fmajor, fminor, Rz, RzT);
}

void run(topo_t *topo, topo_t *awp)
{
        for(int iter = 0; iter < nt; ++iter) {

               topo_stress_interior_H(topo);

	       dstrqc_H_new(awp->xx, awp->yy, awp->zz, awp->xy, awp->xz, awp->yz,
	        	awp->r1, awp->r2, awp->r3, awp->r4, awp->r5, awp->r6,
	        	awp->u1, awp->v1, awp->w1, awp->lami,
	        	awp->mui, awp->qpi, awp->coeff, awp->qsi, awp->dcrjx, awp->dcrjy, awp->dcrjz,
	        	ny,  nz,  awp->stream_1, awp->lam_mu,
	        	awp->vx1, awp->vx2, awp->ww, awp->wwo,
	        	nx, 0,  coord[0], coord[1],   ngsl + 2,  nx + ngsl2 - 1,
	        	 2 + ngsl,  ny + ngsl2 - 1, 0);

        }
}

int compare(topo_t *topo, topo_t *awp)
{

        prec *a[3] = {awp->xx, awp->r1, awp->w1};
        prec *b[3] = {topo->xx, topo->r1, topo->w1};
        const char *names[3] = {"sxx", "r1", "vz"};
        double err[3];
        int nxt = nx - ngsl;
        int nyt = ny - ngsl;
        int nzt = 50;
        double total_error = 0;
        int excl = 4;
        int i0 = excl + ngsl + 2;
        int in = i0 + nxt;
        int j0 = excl + ngsl + 2;
        int jn = j0 + nyt;
        int nbnd = 8;
        int k0 = align + excl + nbnd;
        int kn = k0 + nzt;
        int size = (in - i0) * (jn - j0) * (kn - k0);
        printf("Comparing in region [%d %d %d] [%d %d %d], size = %d \n", i0, j0, k0,
                        in, jn, kn,  size);
        for (int i = 0; i < 2; ++i) {
             err[i] = check_flinferr(a[i], b[i], 
                             i0, in, j0, jn, k0, kn,
                             topo->line, topo->slice);
                printf("%s: %g ", names[i], err[i]);
                total_error += err[i];
        }
        printf("\n");

        topo_h_free(&reference);
        return total_error > 1e-6;
}
