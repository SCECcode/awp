#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define APPLY_BC 0
#define ERROR_TOLERANCE 2.0
#include <topography/topography.h>
#include <topography/metrics/metrics.h>
#include <topography/velocity.cuh>
#include <topography/stress.cuh>
#include <topography/geometry.h>
#include <topography/mapping.h>
#include <grid/shift.h>
#include "functions.c"
#include "grid_check.c"
#include "mms.c"

using _prec=float;

void geom_mapping_z(_prec *out, const fcn_grid_t grid, const int3_t shift,
                    const f_grid_t *metrics_f,
                    const g_grid_t *metrics_g) {
        _prec *g;
        if (shift.z == 0) {
                g = metrics_g->g;
        }
        else {
                g = metrics_g->g_c;
        }

        int3_t nodes = grid_node();
        int3_t u1 = grid_u1();
        int3_t u2 = grid_u2();
        _prec *f;
        if (shift.x == nodes.x && shift.y == nodes.y) {
                f = metrics_f->f;
        } 
        else if(shift.x == u1.x && shift.y == u1.y) {
                f = metrics_f->f_1;
        }
        else if(shift.x == u2.x && shift.y == u2.y) {
                f = metrics_f->f_2;
        }
        else {
                f = metrics_f->f_c;
        }

        int f_offset_x = metrics_f->offset[0] + metrics_f->bounds_stress_x[0];
        int f_offset_y = metrics_f->offset[1] + metrics_f->bounds_stress_y[0];

        // Error: `grid` cannot be larger than the stress grid.
        //assert(f_offset_x + grid.size.x <= metrics_f->mem[0]);
        //assert(f_offset_y + grid.size.y <= metrics_f->mem[1]);

        for (int i = 0; i < grid.size.x; ++i) {
        for (int j = 0; j < grid.size.y; ++j) {
        for (int k = 0; k < grid.size.z; ++k) {
                int pos = grid.offset1.z + k +
                          (grid.offset1.y + j) * grid.line +
                          (grid.offset1.x + i) * grid.slice;
                int pos_g = k + metrics_g->offset;
                int pos_f = f_offset_y + j +
                            (i + f_offset_x) * metrics_f->slice;
                if (k >= MAPPING_START_POINT)
                out[pos] = g[pos_g] * f[pos_f];
                else
                out[pos] = g[pos_g];

        }
        }
        }
}


typedef struct
{
        _prec *vx;
        _prec *vy;
        _prec *vz;
        _prec *sxx;
        _prec *syy;
        _prec *szz;
        _prec *sxy;
        _prec *sxz;
        _prec *syz;
        _prec *rho;
        _prec *lami;
        _prec *mui;
        _prec *qpi;
        _prec *qsi;
        _prec *r1;
        _prec *r2;
        _prec *r3;
        _prec *r4;
        _prec *r5;
        _prec *r6;
        _prec *wwo;
        _prec *vx1, *vx2, *coeff;
        int *ww;
        int num_bytes;
} variables_t;

typedef struct
{
        _prec tol;
        _prec grid_spacing;
        int write_vtk;
        int verbose;
        int num_bytes;
        int3_t size;
        int3_t coord3;
        topo_t T;
        variables_t input;
        variables_t output;
        variables_t answer;
        _prec mms_wavenumber;
} testdata_t;

typedef struct
{
        // parameter coordinates
        _prec *x, *y, *z;
        // physical coordinate
        _prec *zp;
} grid_t;

typedef struct
{
        _prec interior;
        _prec boundary[TOP_BOUNDARY_SIZE];
} err_t;

typedef struct
{
        err_t vx;
        err_t vy;
        err_t vz;
        err_t sxx;
        err_t syy;
        err_t szz;
        err_t sxy;
        err_t sxz;
        err_t syz;
} vars_err_t;


int3_t refine(const int3_t initial_size, const int grid);
prec max_error(vars_err_t *err, const int num_refinements);
void convergence_rates(vars_err_t *rates, const vars_err_t *err, const _prec *h,
                       const int num_refinements);
_prec convergence_rate(const _prec err1, const _prec err2, const _prec h1, const _prec h2);
void test_initialize(testdata_t *test, const int grid, const char *topoography_dir);
void test_velocity(testdata_t *test, vars_err_t *err);
void test_stress(testdata_t *test, vars_err_t *err);
void test_free(testdata_t *test);
void vars_init(variables_t *vars, const int num_bytes);
void vars_copy_to_device(topo_t *topo, const variables_t *vars);
void vars_copy_to_host(variables_t *vars, const topo_t *topo);
void vars_free(variables_t *vars);
void test_grid_data_init(grid_t *data, const testdata_t *test, const fcn_grid_t grid,
                    const int3_t shift);
void test_grid_data_free(grid_t *data);
err_t check_answer(const _prec *u, const _prec *v, const fcn_grid_t grid);
void init_sponge(topo_t *topo, const int num_bytes);


int main(int argc, char **argv)
{
        int num_refinements = 4;

        testdata_t test;
        int3_t initial_size = {16, 16, 16};

        vars_err_t err[num_refinements];
        int grid_sizes[num_refinements];
        _prec grid_spacings[num_refinements];

        const char *topography_dir = argv[1];

        printf("Convergence rate test\n");
        printf("-----------------------------------------------------\n");
        for (int grid = 0; grid < num_refinements; ++grid) {
                test.size = refine(initial_size, grid);
                grid_sizes[grid] = test.size.x;
                test_initialize(&test, grid, topography_dir);
                grid_spacings[grid] = test.grid_spacing;
                printf("Grid refinement: %d  grid size: {%d, %d, %d} \n", 
                        grid, test.size.x, test.size.y, test.size.z);
                test_velocity(&test, &err[grid]);
                printf("Testing stresses\n");
                test_free(&test);
                test_initialize(&test, grid, topography_dir);
                test_stress(&test, &err[grid]);
                test_free(&test);
        }
        printf("-----------------------------------------------------\n");

        vars_err_t rates[num_refinements - 1]; 
        for (int i = 0; i < num_refinements - 1; ++i) {
                convergence_rates(rates, err, grid_spacings, num_refinements); 
        }

        const int show_velocity = 1;
        const int show_stress = 1;


        if (show_velocity) {
        printf("Interior truncation errors\n");
        printf("N \t vx        \t vy          \t vz \n");
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i],
                       err[i].vx.interior, err[i].vy.interior,
                       err[i].vz.interior);
        }
        
        
        printf("Interior error rates\n");
        printf("N \t vx        \t vy          \t vz \n");
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i+1],
                       rates[i].vx.interior, rates[i].vy.interior,
                       rates[i].vz.interior);
        }
        printf("\n");
        printf("\n");
        printf("Boundary truncation errors\n");
        printf("N \t z \t  vx        \t vy          \t vz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %d \t %e \t %e \t %e \n", grid_sizes[i], j , 
                       err[i].vx.boundary[j], err[i].vy.boundary[j],
                       err[i].vz.boundary[j]);
        }
        printf("\n");
        }

        
        printf("\n");
        printf("Boundary error rates\n");
        printf("N \t z \t vx        \t vy          \t vz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %d \t %e \t %e \t %e \n", grid_sizes[i+1], j,
                       rates[i].vx.boundary[j], rates[i].vy.boundary[j],
                       rates[i].vz.boundary[j]);
        }
        printf("\n");
        }


        }

        if (show_stress) {
        printf("Interior truncation error\n");
        printf("N \t sxx        \t syy          \t szz \n");
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i],
                       err[i].sxx.interior, err[i].syy.interior,
                       err[i].szz.interior);
        }

        printf("N \t sxy        \t sxz          \t syz \n");
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i],
                       err[i].sxy.interior, err[i].sxz.interior,
                       err[i].syz.interior);
        }
        printf("\n");
        printf("\n");

        printf("Interior error rates\n");


        printf("N \t sxx        \t syy          \t szz \n");
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i+1],
                       rates[i].sxx.interior, rates[i].syy.interior,
                       rates[i].szz.interior);
        }

        printf("N \t sxy        \t sxz          \t syz \n");
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i+1],
                       rates[i].sxy.interior, rates[i].sxz.interior,
                       rates[i].syz.interior);
        }

        printf("\n");
        printf("\n");

        printf("Boundary truncation errors\n");
        
        printf("N \t z \t sxx        \t syy          \t szz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %d \t %e \t %e \t %e \n", grid_sizes[i], j,
                       err[i].sxx.boundary[j], err[i].syy.boundary[j],
                       err[i].szz.boundary[j]);
        }
        printf("\n");
        }

        printf("N \t sxy        \t sxz          \t syz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements; ++i) {
                printf("%d \t %e \t %e \t %e \n", grid_sizes[i],
                       err[i].sxy.boundary[j], err[i].sxz.boundary[j],
                       err[i].syz.boundary[j]);
        }
        printf("\n");
        }

        printf("Boundary error rates\n");


        printf("N \t z \t sxx        \t syy          \t szz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %d \t %e \t %e \t %e \n", grid_sizes[i+1], j,
                       rates[i].sxx.boundary[j], rates[i].syy.boundary[j],
                       rates[i].szz.boundary[j]);
        }
        printf("\n");
        }

        printf("N \t z \t sxy        \t sxz          \t syz \n");
        for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
        for (int i = 0; i < num_refinements - 1; ++i) {
                printf("%d \t %d \t %e \t %e \t %e \n", grid_sizes[i+1], j,
                       rates[i].sxy.boundary[j], rates[i].sxz.boundary[j],
                       rates[i].syz.boundary[j]);
        }
        printf("\n");
        }


        }



        return !(max_error(err, num_refinements) < ERROR_TOLERANCE);
}

int3_t refine(const int3_t initial_size, const int grid) 
{
        int3_t out;
        out.x = initial_size.x*pow(2, grid);
        out.y = initial_size.y*pow(2, grid);
        out.z = initial_size.z*pow(2, grid);
        return out;
}

void convergence_rates(vars_err_t *rates, const vars_err_t *err, const _prec *h,
                       const int num_refinements) {
        for (int i = 0; i < num_refinements - 1; ++i) {
                rates[i].vx.interior = convergence_rate(
                    err[i].vx.interior, err[i + 1].vx.interior, h[i], h[i + 1]);
                rates[i].vy.interior = convergence_rate(
                    err[i].vy.interior, err[i + 1].vy.interior, h[i], h[i + 1]);
                rates[i].vz.interior = convergence_rate(
                    err[i].vz.interior, err[i + 1].vz.interior, h[i], h[i + 1]);
                rates[i].sxx.interior =
                    convergence_rate(err[i].sxx.interior,
                                     err[i + 1].sxx.interior, h[i], h[i + 1]);
                rates[i].syy.interior =
                    convergence_rate(err[i].syy.interior,
                                     err[i + 1].syy.interior, h[i], h[i + 1]);
                rates[i].szz.interior =
                    convergence_rate(err[i].szz.interior,
                                     err[i + 1].szz.interior, h[i], h[i + 1]);
                rates[i].sxy.interior =
                    convergence_rate(err[i].sxy.interior,
                                     err[i + 1].sxy.interior, h[i], h[i + 1]);
                rates[i].sxz.interior =
                    convergence_rate(err[i].sxz.interior,
                                     err[i + 1].sxz.interior, h[i], h[i + 1]);
                rates[i].syz.interior =
                    convergence_rate(err[i].syz.interior,
                                     err[i + 1].syz.interior, h[i], h[i + 1]);

                for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
                rates[i].vx.boundary[j] = convergence_rate(
                    err[i].vx.boundary[j], err[i + 1].vx.boundary[j], h[i], h[i + 1]);
                rates[i].vy.boundary[j] = convergence_rate(
                    err[i].vy.boundary[j], err[i + 1].vy.boundary[j], h[i], h[i + 1]);
                rates[i].vz.boundary[j] = convergence_rate(
                    err[i].vz.boundary[j], err[i + 1].vz.boundary[j], h[i], h[i + 1]);
                rates[i].sxx.boundary[j] =
                    convergence_rate(err[i].sxx.boundary[j],
                                     err[i + 1].sxx.boundary[j], h[i], h[i + 1]);
                rates[i].syy.boundary[j] =
                    convergence_rate(err[i].syy.boundary[j],
                                     err[i + 1].syy.boundary[j], h[i], h[i + 1]);
                rates[i].szz.boundary[j] =
                    convergence_rate(err[i].szz.boundary[j],
                                     err[i + 1].szz.boundary[j], h[i], h[i + 1]);
                rates[i].sxy.boundary[j] =
                    convergence_rate(err[i].sxy.boundary[j],
                                     err[i + 1].sxy.boundary[j], h[i], h[i + 1]);
                rates[i].sxz.boundary[j] =
                    convergence_rate(err[i].sxz.boundary[j],
                                     err[i + 1].sxz.boundary[j], h[i], h[i + 1]);
                rates[i].syz.boundary[j] =
                    convergence_rate(err[i].syz.boundary[j],
                                     err[i + 1].syz.boundary[j], h[i], h[i + 1]);
                }
        }
}


prec max_error(vars_err_t *err, const int num_refinements) {

                double err_max = 0.0;
                for (int i = 0; i < num_refinements - 1; ++i) {
                for (int j = 0; j < TOP_BOUNDARY_SIZE; ++j) {
                    err_max = max(err_max, err[i].vx.boundary[j]);
                    err_max = max(err_max, err[i].vy.boundary[j]);
                    err_max = max(err_max, err[i].vz.boundary[j]);
                    err_max = max(err_max, err[i].sxx.boundary[j]);
                    err_max = max(err_max, err[i].syy.boundary[j]);
                    err_max = max(err_max, err[i].szz.boundary[j]);
                    err_max = max(err_max, err[i].sxy.boundary[j]);
                    err_max = max(err_max, err[i].sxz.boundary[j]);
                    err_max = max(err_max, err[i].syz.boundary[j]);
                }
                }
                return err_max;

}

_prec convergence_rate(const _prec err1, const _prec err2, const _prec h1,
                       const _prec h2) {
        return log(err1/err2)/log(h1/h2);
}

void test_initialize(testdata_t *test, const int grid, const char *topography_dir)
{
        int rank = 0;
        int x_rank_l = -1;
        int x_rank_r = -1;
        int y_rank_f = -1;
        int y_rank_b = -1;
        int coord[2] = {0, 0};
        int px = 1;
        int py = 1;
        hipStream_t stream_1, stream_2, stream_i;
        hipStreamCreate(&stream_1);
        hipStreamCreate(&stream_2);
        hipStreamCreate(&stream_i);
        test->tol = 1e-6;
        _prec dt = 1.0;
        _prec h  = 1.0/(test->size.x - 2 - OVERLAP);
        printf("Test size: %d %d %d \n", test->size.x, test->size.y, test->size.z);
        char gridname[2048];
        sprintf(gridname, "%s/topography_%d.bin", topography_dir, grid);
        test->T = topo_init(1, gridname, rank, x_rank_l, x_rank_r, y_rank_f,
                            y_rank_b, coord, px, py, test->size.x, test->size.y,
                            test->size.z, dt, h, h, h, stream_1, stream_2, stream_i);
        test->T.timestep = 0;
        topo_d_malloc(&test->T);
        test->coord3.x = coord[0];
        test->coord3.y = coord[1];
        test->grid_spacing = h;
        test->write_vtk = 0;
        test->mms_wavenumber = 2 * M_PI * 4;

        _prec amplitude = 0.0;
        _prec3_t width = {.x = 0.1, .y = 0.1, .z = 0};
        _prec3_t center = {.x = 0.5, .y = 0.5, .z = 0};

        topo_init_metrics(&test->T);
        topo_init_geometry(&test->T);
        topo_build(&test->T);
        topo_set_constants(&test->T);

        int num_items = test->T.mx*test->T.my*test->T.mz;
        vars_init(&test->input, num_items);
        vars_init(&test->output,num_items);
        vars_init(&test->answer,num_items);

        init_sponge(&test->T, sizeof(_prec)*num_items);
}

void test_velocity(testdata_t *test, vars_err_t *err)
{

        int3_t shift = {0, 0, 0};
        fcn_grid_t velocity_grid = fcn_init_grid(
            test->size, shift, test->coord3, 0, test->grid_spacing);
        fcn_grid_t stress_grid = fcn_init_grid(test->size, shift, test->coord3,
                                               ngsl / 2, test->grid_spacing);
        grid_t gvx;
        grid_t gvy;
        grid_t gvz;
        grid_t gsxx;
        grid_t gsyy;
        grid_t gszz;
        grid_t gsxy;
        grid_t gsxz;
        grid_t gsyz;
        test_grid_data_init(&gvx,  test, stress_grid, grid_u1());  
        test_grid_data_init(&gvy,  test, stress_grid, grid_u2());  
        test_grid_data_init(&gvz,  test, stress_grid, grid_u3());  
        test_grid_data_init(&gsxx, test, stress_grid, grid_xx());  
        test_grid_data_init(&gsyy, test, stress_grid, grid_yy());  
        test_grid_data_init(&gszz, test, stress_grid, grid_zz());  
        test_grid_data_init(&gsxy, test, stress_grid, grid_xy());  
        test_grid_data_init(&gsxz, test, stress_grid, grid_xz());  
        test_grid_data_init(&gsyz, test, stress_grid, grid_yz());  


        // Input
        _prec properties[2] = {test->mms_wavenumber, 0};
        fcn_apply(test->input.sxx, mms_init_sxx, gsxx.x, gsxx.y, gsxx.zp,
                  properties, stress_grid);
        fcn_apply(test->input.syy, mms_init_syy, gsyy.x, gsyy.y, gsyy.zp,
                  properties, stress_grid);
        fcn_apply(test->input.szz, mms_init_szz, gszz.x, gszz.y, gszz.zp,
                  properties, stress_grid);
        fcn_apply(test->input.sxy, mms_init_sxy, gsxy.x, gsxy.y, gsxy.zp,
                  properties, stress_grid);
        fcn_apply(test->input.sxz, mms_init_sxz, gsxz.x, gsxz.y, gsxz.zp,
                  properties, stress_grid);
        fcn_apply(test->input.syz, mms_init_syz, gsyz.x, gsyz.y, gsyz.zp,
                  properties, stress_grid);

        vars_copy_to_device(&test->T, &test->input);

        topo_velocity_interior_H(&test->T);

        // Output
        vars_copy_to_host(&test->output, &test->T);
        
        //Check answer
        fcn_apply(test->answer.vx, mms_final_vx, gvx.x, gvx.y, gvx.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.vy, mms_final_vy, gvy.x, gvy.y, gvy.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.vz, mms_final_vz, gvz.x, gvz.y, gvz.zp,
                  properties, velocity_grid);


        err_t tmp = check_answer(test->output.vx, test->answer.vx, velocity_grid);
        err->vx = tmp;

        tmp = check_answer(test->output.vy, test->answer.vy, velocity_grid);
        err->vy = tmp;

        tmp = check_answer(test->output.vz, test->answer.vz, velocity_grid);
        err->vz = tmp;
        // Exclude solution at ghost point
        err->vz.boundary[TOP_BOUNDARY_SIZE-1] = 0.0;

        char vtk_file[128];
        if (test->write_vtk) {
                sprintf(vtk_file, "input_sxx.vtk");
                vtk_write_grid(vtk_file, gsxx.x, gsxx.y, gsxx.zp,
                               velocity_grid);
                vtk_append_scalar(vtk_file, "z", test->input.sxx,
                                  velocity_grid);

                sprintf(vtk_file, "output_vx.vtk");
                vtk_write_grid(vtk_file, gvx.x, gvx.y, gvx.zp, velocity_grid);
                vtk_append_scalar(vtk_file, "z", test->output.vx,
                                  velocity_grid);

                sprintf(vtk_file, "answer_vx.vtk");
                vtk_write_grid(vtk_file, gvx.x, gvx.y, gvx.zp, velocity_grid);
                vtk_append_scalar(vtk_file, "z", test->answer.vx,
                                  velocity_grid);
        }
}

void test_stress(testdata_t *test, vars_err_t *err)
{

        int3_t shift = {0, 0, 0};
        fcn_grid_t velocity_grid = fcn_init_grid(
            test->size, shift, test->coord3, 0, test->grid_spacing);
        fcn_grid_t stress_grid = fcn_init_grid(test->size, shift, test->coord3,
                                               ngsl / 2, test->grid_spacing);
        grid_t gvx;
        grid_t gvy;
        grid_t gvz;
        grid_t gsxx;
        grid_t gsyy;
        grid_t gszz;
        grid_t gsxy;
        grid_t gsxz;
        grid_t gsyz;
        test_grid_data_init(&gvx,  test, stress_grid, grid_u1());  
        test_grid_data_init(&gvy,  test, stress_grid, grid_u2());  
        test_grid_data_init(&gvz,  test, stress_grid, grid_u3());  
        test_grid_data_init(&gsxx, test, stress_grid, grid_xx());  
        test_grid_data_init(&gsyy, test, stress_grid, grid_yy());  
        test_grid_data_init(&gszz, test, stress_grid, grid_zz());  
        test_grid_data_init(&gsxy, test, stress_grid, grid_xy());  
        test_grid_data_init(&gsxz, test, stress_grid, grid_xz());  
        test_grid_data_init(&gsyz, test, stress_grid, grid_yz());  

        // Input
        _prec properties[2] = {test->mms_wavenumber, 0};
        fcn_apply(test->input.vx, mms_init_vx, gvx.x, gvx.y, gvx.zp,
                  properties, stress_grid);
        fcn_apply(test->input.vy, mms_init_vy, gvy.x, gvy.y, gvy.zp,
                  properties, stress_grid);
        fcn_apply(test->input.vz, mms_init_vz, gvz.x, gvz.y, gvz.zp,
                  properties, stress_grid);
        
        vars_copy_to_device(&test->T, &test->input);

        topo_stress_interior_H(&test->T);

        // Output
        vars_copy_to_host(&test->output, &test->T);

        // Answer
        fcn_apply(test->answer.sxx, mms_final_sxx, gsxx.x, gsxx.y, gsxx.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.syy, mms_final_syy, gsyy.x, gsyy.y, gsyy.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.szz, mms_final_szz, gszz.x, gszz.y, gszz.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.sxy, mms_final_sxy, gsxy.x, gsxy.y, gsxy.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.sxz, mms_final_sxz, gsxz.x, gsxz.y, gsxz.zp,
                  properties, velocity_grid);
        fcn_apply(test->answer.syz, mms_final_syz, gsyz.x, gsyz.y, gsyz.zp,
                  properties, velocity_grid);


        // Exclude solution at ghost point
        err->vz.boundary[TOP_BOUNDARY_SIZE-1] = 0.0;
        err_t tmp;
        tmp = check_answer(test->output.sxx, test->answer.sxx, velocity_grid);
        err->sxx = tmp;
        tmp = check_answer(test->output.syy, test->answer.syy, velocity_grid);
        err->syy = tmp;
        tmp = check_answer(test->output.szz, test->answer.szz, velocity_grid);
        err->szz = tmp;
        tmp = check_answer(test->output.sxy, test->answer.sxy, velocity_grid);
        err->sxy = tmp;
        tmp = check_answer(test->output.sxz, test->answer.sxz, velocity_grid);
        err->sxz = tmp;
        tmp = check_answer(test->output.syz, test->answer.syz, velocity_grid);
        err->syz = tmp;

        // Exclude solution at ghost point
        err->sxz.boundary[TOP_BOUNDARY_SIZE-1] = 0.0;
        err->syz.boundary[TOP_BOUNDARY_SIZE-1] = 0.0;
}

void test_free(testdata_t *test)
{
        topo_free(&test->T);
        hipStreamDestroy(test->T.stream_1);
        hipStreamDestroy(test->T.stream_2);
        hipStreamDestroy(test->T.stream_i);
        vars_free(&test->input);
        vars_free(&test->output);
        vars_free(&test->answer);
}

void vars_init(variables_t *vars, const int num_items)
{
        int item_size = sizeof(_prec);
        vars->vx = (_prec*)calloc(num_items, item_size);
        vars->vy = (_prec*)calloc(num_items, item_size);
        vars->vz = (_prec*)calloc(num_items, item_size);
        vars->sxx =(_prec*) calloc(num_items, item_size);
        vars->syy =(_prec*) calloc(num_items, item_size);
        vars->szz =(_prec*) calloc(num_items, item_size);
        vars->sxy =(_prec*) calloc(num_items, item_size);
        vars->sxz =(_prec*) calloc(num_items, item_size);
        vars->syz =(_prec*) calloc(num_items, item_size);
        vars->rho =(_prec*) calloc(num_items, item_size);
        vars->lami =(_prec*) calloc(num_items, item_size);
        vars->mui =(_prec*) calloc(num_items, item_size);
        vars->qpi =(_prec*) calloc(num_items, item_size);
        vars->qsi =(_prec*) calloc(num_items, item_size);
        vars->r1 =(_prec*) calloc(num_items, item_size);
        vars->r2 =(_prec*) calloc(num_items, item_size);
        vars->r3 =(_prec*) calloc(num_items, item_size);
        vars->r4 =(_prec*) calloc(num_items, item_size);
        vars->r5 =(_prec*) calloc(num_items, item_size);
        vars->r6 =(_prec*) calloc(num_items, item_size);
        vars->wwo =(_prec*) calloc(num_items, item_size);
        vars->vx1 =(_prec*) calloc(num_items, item_size);
        vars->vx2 =(_prec*) calloc(num_items, item_size);
        vars->coeff =(_prec*) calloc(num_items, item_size);
        vars->ww =(int*) calloc(num_items, item_size);
        vars->num_bytes = num_items*item_size;
        for (int i = 0; i < num_items; ++i) {
            vars->rho[i] = 1.0;
            vars->lami[i] = 1.0;
            vars->mui[i] = 1.0;
            vars->ww[i] = 1;
            vars->wwo[i] = 1.0;

        }
}

void vars_copy_to_device(topo_t *topo, const variables_t *vars)
{
        hipMemcpy(topo->u1, vars->vx, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->v1, vars->vy, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->w1, vars->vz, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->xx, vars->sxx, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->yy, vars->syy, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->zz, vars->szz, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->xy, vars->sxy, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->xz, vars->sxz, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->yz, vars->syz, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->rho, vars->rho, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->lami, vars->lami, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->mui, vars->mui, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->qpi, vars->qpi, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->qsi, vars->qsi, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r1, vars->r1, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r2, vars->r2, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r3, vars->r3, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r4, vars->r4, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r5, vars->r5, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->r6, vars->r6, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->wwo, vars->wwo, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->vx1, vars->vx1, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->vx2, vars->vx2, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->coeff, vars->coeff, vars->num_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(topo->ww, vars->ww, vars->num_bytes,
                   hipMemcpyHostToDevice);
}

void vars_copy_to_host(variables_t *vars, const topo_t *topo)
{
        hipMemcpy(vars->vx, topo->u1, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->vy, topo->v1, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->vz, topo->w1, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->sxx, topo->xx, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->syy, topo->yy, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->szz, topo->zz, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->sxy, topo->xy, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->sxz, topo->xz, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->syz, topo->yz, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->rho, topo->rho, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->lami, topo->lami, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->mui, topo->mui, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->qpi, topo->qpi, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->qsi, topo->qsi, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r1, topo->r1, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r2, topo->r2, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r3, topo->r3, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r4, topo->r4, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r5, topo->r5, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->r6, topo->r6, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->wwo, topo->wwo, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->vx1, topo->vx1, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->vx2, topo->vx2, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->coeff, topo->coeff, vars->num_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(vars->ww, topo->ww, vars->num_bytes,
                   hipMemcpyDeviceToHost);
}
void vars_free(variables_t *vars)
{
        free(vars->vx);
        free(vars->vy);
        free(vars->vz);
        free(vars->sxx);
        free(vars->syy);
        free(vars->szz);
        free(vars->sxy);
        free(vars->sxz);
        free(vars->syz);
        free(vars->rho);
        free(vars->lami);
        free(vars->mui);
        free(vars->qpi);
        free(vars->qsi);
        free(vars->r1);
        free(vars->r2);
        free(vars->r3);
        free(vars->r4);
        free(vars->r5);
        free(vars->r6);
}

void test_grid_data_init(grid_t *data, const testdata_t *test, const fcn_grid_t grid,
                    const int3_t shift) {
        data->x = (_prec*)malloc(grid.num_bytes);
        data->y = (_prec*)malloc(grid.num_bytes);
        data->z = (_prec*)malloc(grid.num_bytes);
        data->zp =(_prec*) malloc(grid.num_bytes);

        fcn_fill_grid(data->x, grid, shift, 0);
        fcn_fill_grid(data->y, grid, shift, 1);
        fcn_fill_grid(data->z, grid, shift, 2);

        fcn_shift(data->x, data->x, grid, -ngsl*grid.gridspacing);
        fcn_shift(data->y, data->y, grid, -ngsl*grid.gridspacing);

        geom_mapping_z(data->zp, grid, shift, &test->T.metrics_f,
                       &test->T.metrics_g);
}

void test_grid_data_free(grid_t *data)
{
        free(data->x);
        free(data->y);
        free(data->z);
        free(data->zp);
}

err_t check_answer(const _prec *u, const _prec *v, const fcn_grid_t grid)
{
        // Maximum truncation error at the boundary points
        const int nb = TOP_BOUNDARY_SIZE;
        err_t out;
        for (int i = 0; i < nb; ++i) {
        out.boundary[i] = check_flinferr(u, v, 
                  grid.offset1.x + nb, grid.offset2.x - nb,
                  grid.offset1.y + nb, grid.offset2.y - nb,
                  grid.offset2.z - nb + i, grid.offset2.z - nb + i + 1,
                  grid.line,
                  grid.slice);
        }
        
        // Maximum truncation error in the interior of the domain
        out.interior = check_flinferr(u, v, 
                  grid.offset1.x + nb, grid.offset2.x - nb,
                  grid.offset1.y + nb, grid.offset2.y - nb,
                  grid.offset1.z + nb, 
                  grid.offset2.z - nb - grid.exclude_top_row,
                  grid.line,
                  grid.slice);
        return out;
}

void init_sponge(topo_t *topo, const int num_bytes)
{
        _prec *ones = (_prec*)malloc(num_bytes);
        for (size_t i = 0; i < num_bytes/(sizeof(_prec)); ++i) {
                ones[i] = 1.0;
        }

        hipMemcpy(topo->dcrjx, ones, num_bytes, hipMemcpyHostToDevice);
        hipMemcpy(topo->dcrjy, ones, num_bytes, hipMemcpyHostToDevice);
        hipMemcpy(topo->dcrjz, ones, num_bytes, hipMemcpyHostToDevice);
        free(ones);
}

