#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

#include <topography/sources/source.cuh>
#include <interpolation/interpolation.cuh>
#include <test/test.h>

void cusource_add_cartesian_H(const cu_interp_t *I, prec *out, const prec *in,
                              const prec h, const prec dt)
{
        dim3 block (INTERP_THREADS, 1, 1);
        dim3 grid((I->num_query + INTERP_THREADS - 1) / INTERP_THREADS,
                  1, 1);

        cusource_add_cartesian<<<grid, block>>>(
            out, in, I->d_lx, I->d_ly, I->d_lz, I->num_basis, I->d_ix, I->d_iy,
            I->d_iz, I->d_ridx, h, dt, I->num_query, I->grid);
        CUCHK(hipGetLastError());
}

__global__ void cusource_add_cartesian(prec *out, const prec *in,
                                 const prec *lx, const prec *ly, const prec *lz,
                                 const int num_basis, const int *ix,
                                 const int *iy, const int *iz,
                                 const int *lidx,
                                 const prec h, const prec dt,
                                 const int num_query, const grid3_t grid)
{
        int q = threadIdx.x + blockDim.x * blockIdx.x;
        if (q >= num_query) {
                return;
        }

        prec dth = dt/(h * h * h);

        for (int i = 0; i < num_basis; ++i) {
        for (int j = 0; j < num_basis; ++j) {
        for (int k = 0; k < num_basis; ++k) {
                size_t pos = grid_index(grid, ix[q] + i, iy[q] + j, iz[q] + k);
                out[pos] += - dth * lx[q * num_basis + i] *
                            ly[q * num_basis + j] * lz[q * num_basis + k] *
                            in[lidx[q]];
        }
        }
        }
}

void cusource_add_curvilinear_H(const cu_interp_t *I, prec *out, const prec *in,
                                const prec h, const prec dt, const prec *f,
                                const int ny, const prec *dg) 
{
        dim3 block (INTERP_THREADS, 1, 1);
        dim3 grid((I->num_query + INTERP_THREADS - 1) / INTERP_THREADS,
                  1, 1);

        cusource_add_curvilinear<<<grid, block>>>(
            out, in, I->d_lx, I->d_ly, I->d_lz, I->num_basis, I->d_ix, I->d_iy,
            I->d_iz, I->d_ridx, h, dt, I->num_query, I->grid, f, ny, dg);
        CUCHK(hipGetLastError());
}

__global__ void cusource_add_curvilinear(prec *out, const prec *in,
                                 const prec *lx, const prec *ly, const prec *lz,
                                 const int num_basis, const int *ix,
                                 const int *iy, const int *iz,
                                 const int *lidx,
                                 const prec h, const prec dt,
                                 const int num_query, const grid3_t grid,
                                 const prec *f, const int ny, const prec *dg)
{
        int q = threadIdx.x + blockDim.x * blockIdx.x;
        if (q >= num_query) {
                return;
        }

#define _f(i, j)                                                             \
  f[(j) + align +                                                     \
      ((i) + 2) * (2 * align + 2 * ngsl + ny + 4) + 2]
#define _dg(k) dg[(k) + align]

        prec dth = dt / (h * h * h);

        for (int i = 0; i < num_basis; ++i) {
        for (int j = 0; j < num_basis; ++j) {
        for (int k = 0; k < num_basis; ++k) {
               prec Ji =
                   1.0 / (_f(i + ix[q], j + iy[q]) *
                          _dg(iz[q] + k));
                size_t pos = grid_index(grid, ix[q] + i, iy[q] + j, iz[q] + k);
                out[pos] += - dth * lx[q * num_basis + i] *
                            ly[q * num_basis + j] * lz[q * num_basis + k] *
                            in[lidx[q]] * Ji;
        }
        }
        }
}

void cusource_add_force_H(const cu_interp_t *I, prec *out, const prec *in,
                          const prec *d1, const prec h, const prec dt,
                          const prec quad_weight,
                          const prec *f, const int nx, const int ny,
                          const int nz, const prec *dg) 
{
        dim3 block (INTERP_THREADS, 1, 1);
        dim3 grid((I->num_query + INTERP_THREADS - 1) / INTERP_THREADS,
                  1, 1);

        cusource_add_force<<<grid, block>>>(
            out, in, d1, I->d_lx, I->d_ly, I->d_lz, I->num_basis, I->d_ix,
            I->d_iy, I->d_iz, I->d_ridx, h, dt, quad_weight, I->num_query,
            I->grid, f, nx, ny, nz, dg);
        CUCHK(hipGetLastError());
}

__global__ void cusource_add_force(prec *out, const prec *in, const prec *d1,
                                   const prec *lx, const prec *ly,
                                   const prec *lz, const int num_basis,
                                   const int *ix, const int *iy, const int *iz,
                                   const int *lidx, const prec h, const prec dt,
                                   const prec quad_weight,
                                   const int num_query, const grid3_t grid,
                                   const prec *f, const int nx, const int ny,
                                   const int nz, const prec *dg) 
{
        int q = threadIdx.x + blockDim.x * blockIdx.x;
        if (q >= num_query) {
                return;
        }

#define _f(i, j)                                                             \
  f[(j) + align +                                                     \
      ((i) + 2) * (2 * align + 2 * ngsl + ny + 4) + 2]
#define _dg(k) dg[(k) + align]

#define _rho(i, j, k)                                                  \
        d1[(k) + align +                                               \
           (2 * align + nz) * ((i) + ngsl + 2) * (2 * ngsl + ny + 4) + \
           (2 * align + nz) * ((j) + ngsl + 2)]

        prec dth = dt / (h * h * h);

        for (int i = 0; i < num_basis; ++i) {
        for (int j = 0; j < num_basis; ++j) {
        for (int k = 0; k < num_basis; ++k) {
                prec Ji =
                    - quad_weight / (_f(i + ix[q], j + iy[q]) * _dg(iz[q] + k) *
                                   _rho(i + ix[q], j + iy[q], iz[q] + k));
                size_t pos = grid_index(grid, ix[q] + i, iy[q] + j, iz[q] + k);
                out[pos] += -dth * lx[q * num_basis + i] *
                            ly[q * num_basis + j] * lz[q * num_basis + k] * in[lidx[q]] * Ji;
        }
        }
        }
}

