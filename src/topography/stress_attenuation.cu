#include <hip/hip_runtime.h>
#include <nvToolsExt.h>
#include <stdio.h>

#include <topography/kernels/stress_attenuation.cuh>
#include <topography/kernels/optimized_launch_config.cuh>
#include <topography/stress_attenuation.cuh>
#include <test/test.h>


// Threads in x, y, z
#ifndef STRIU_TX
#define STRIU_TX 32
#endif      
            
#ifndef STRIU_TY
#define STRIU_TY 1
#endif      
            
#ifndef STRIU_TZ
#define STRIU_TZ 4
#endif

// Unroll factor in CUDA x
#ifndef STRIU_RX
#define STRIU_RX 1
#endif

// Unroll factor in CUDA y
#ifndef STRIU_RY
#define STRIU_RY 2
#endif

#include "kernels/stress_attenuation.cu"
#include "kernels/stress_index_unroll.cuh"

inline dim3 set_grid(const dim3 block, const int3_t size, const dim3 loop)
{
        dim3 out;
        out.x = ((1 - loop.x) * size.z + block.x - 1 + loop.x) / block.x;
        out.y = ((1 - loop.y) * size.y + block.y - 1 + loop.y) / block.y;
        out.z = ((1 - loop.z) * size.x + block.z - 1 + loop.z) / block.z;
        return out;
}

void topo_set_constants(topo_t *T)
{
        set_constants(T->gridspacing, T->dth * T->gridspacing, T->nx, T->ny,
                      T->nz);
}

void topo_stress_interior_H(topo_t *T)
{

        if (!T->use) return;
        if (TOPO_DBG) {
                printf("launching %s(%d)\n", __func__, T->rank);
        }


     int shift = ngsl + 2;
     {
     int3_t size = {T->stress_bounds_right[0] - T->stress_bounds_left[0], 
                    T->stress_bounds_ydir[1] -  T->stress_bounds_ydir[0],
                    (int)T->stress_grid_interior.z};

        dim3 threads (STRIU_TX, STRIU_TY, STRIU_TZ);
        dim3 blocks((size.z - 4) / (STRIU_RX * threads.x) + 1,
                    (size.y - 1) / (STRIU_RY * threads.y) + 1,
                    (size.x - 1) / (threads.z) + 1);

        dtopo_str_111_index_unroll<STRIU_TX, STRIU_TY, STRIU_TZ, STRIU_RX, STRIU_RY><<<blocks, threads, 0, T->stream_i>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_left[1] + shift, 
                          T->stress_bounds_right[0]+ shift, 
                          T->stress_bounds_ydir[0] + shift, 
                          T->stress_bounds_ydir[1] + shift);

        CUCHK(hipGetLastError());
        }

        {
        dim3 block(DTOPO_STR_112_X, DTOPO_STR_112_Y,
                    DTOPO_STR_112_Z);
        int3_t size = {(int)T->stress_bounds_right[0] - T->stress_bounds_left[0], 
                       (int)T->stress_bounds_ydir[1] -  T->stress_bounds_ydir[0],
                       TOP_BOUNDARY_SIZE};
        dim3 loop(0, 0, DTOPO_STR_112_LOOP_Z);
        dim3 grid = set_grid(block, size, loop);
        dtopo_str_112<<<grid, block, 0, T->stream_i>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_left[1]  + shift, 
                          T->stress_bounds_right[0] + shift, 
                          T->stress_bounds_ydir[0]  + shift, 
                          T->stress_bounds_ydir[1]  + shift);
        CUCHK(hipGetLastError());
        }
}

void topo_stress_left_H(topo_t *T)
{

        if (!T->use) return;
        if (T->x_rank_l < 0) {
                return;
        }

        if (TOPO_DBG) {
                printf("launching %s(%d)\n", __func__, T->rank);
        }
        int3_t size = {(int)T->stress_bounds_left[1] - T->stress_bounds_left[0],
                       (int)T->stress_bounds_ydir[1] - T->stress_bounds_ydir[0],
                       (int)T->stress_grid_interior.z};

        int shift = ngsl + 2;

        dim3 threads (STRIU_TX, STRIU_TY, STRIU_TZ);
        dim3 blocks((size.z - 4) / (STRIU_RX * threads.x) + 1,
                    (size.y - 1) / (STRIU_RY * threads.y) + 1,
                    (size.x - 1) / (threads.z) + 1);

        dtopo_str_111_index_unroll<STRIU_TX, STRIU_TY, STRIU_TZ, STRIU_RX, STRIU_RY><<<blocks, threads, 0, T->stream_1>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_left[0] + shift, 
                          T->stress_bounds_left[1] + shift, 
                          T->stress_bounds_ydir[0] + shift, 
                          T->stress_bounds_ydir[1] + shift);
        CUCHK(hipGetLastError());


        {
        dim3 block(DTOPO_STR_112_X, DTOPO_STR_112_Y,
                    DTOPO_STR_112_Z);
        int3_t size = {(int)T->stress_bounds_left[1] - T->stress_bounds_left[0],
                       (int)T->stress_bounds_ydir[1] - T->stress_bounds_ydir[0],
                       (int)T->stress_grid_interior.z};
        dim3 loop(0, 0, DTOPO_STR_112_LOOP_Z);
        dim3 grid = set_grid(block, size, loop);
        dtopo_str_112<<<grid, block, 0, T->stream_1>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_left[0] + shift, 
                          T->stress_bounds_left[1] + shift, 
                          T->stress_bounds_ydir[0] + shift, 
                          T->stress_bounds_ydir[1] + shift);
        CUCHK(hipGetLastError());
        }
}

void topo_stress_right_H(topo_t *T)
{

        if (!T->use) return;
        if (T->x_rank_r < 0) {
                return;
        }
        if (TOPO_DBG) {
                printf("launching %s(%d)\n", __func__, T->rank);
        }

        int shift = ngsl + 2;
        {
        int3_t size = {(int)T->stress_bounds_right[1] - T->stress_bounds_left[0],
                       (int)T->stress_bounds_ydir[1] - T->stress_bounds_ydir[0],
                       (int)T->stress_grid_interior.z};

        dim3 threads (STRIU_TX, STRIU_TY, STRIU_TZ);
        dim3 blocks((size.z - 4) / (STRIU_RX * threads.x) + 1,
                    (size.y - 1) / (STRIU_RY * threads.y) + 1,
                    (size.x - 1) / (threads.z) + 1);

        dtopo_str_111_index_unroll<STRIU_TX, STRIU_TY, STRIU_TZ, STRIU_RX, STRIU_RY><<<blocks, threads, 0, T->stream_2>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_right[0] + shift, 
                          T->stress_bounds_right[1] + shift, 
                          T->stress_bounds_ydir[0]  + shift, 
                          T->stress_bounds_ydir[1]  + shift);
        CUCHK(hipGetLastError());
        }

        {
        dim3 block(DTOPO_STR_112_X, DTOPO_STR_112_Y,
                    DTOPO_STR_112_Z);
        int3_t size = {(int)T->stress_bounds_right[1] - T->stress_bounds_left[0],
                       (int)T->stress_bounds_ydir[1] - T->stress_bounds_ydir[0],
                       TOP_BOUNDARY_SIZE};
        dim3 loop(0, 0, DTOPO_STR_112_LOOP_Z);
        dim3 grid = set_grid(block, size, loop);
        dtopo_str_112<<<grid, block, 0, T->stream_2>>>
                         (
                          T->xx, T->yy, T->zz, 
                          T->xy, T->xz, T->yz,
                          T->r1, T->r2, T->r3,
                          T->r4, T->r5, T->r6,
                          T->u1, T->v1, T->w1, 
                          T->metrics_f.d_f,
                          T->metrics_f.d_f1_1,
                          T->metrics_f.d_f1_2,
                          T->metrics_f.d_f1_c,
                          T->metrics_f.d_f2_1,
                          T->metrics_f.d_f2_2,
                          T->metrics_f.d_f2_c,
                          T->metrics_f.d_f_1,
                          T->metrics_f.d_f_2,
                          T->metrics_f.d_f_c,
                          T->metrics_g.d_g,
                          T->metrics_g.d_g3,
                          T->metrics_g.d_g3_c,
                          T->metrics_g.d_g_c,
                          T->lami,
                          T->mui, 
                          T->qpi,
                          T->coeff,
                          T->qsi,
                          T->dcrjx, T->dcrjy, T->dcrjz,
                          T->vx1,
                          T->vx2,
                          T->ww,
                          T->wwo,
                          T->nx, T->ny, T->nz, T->coord[0], T->coord[1], T->nz,
                          T->stress_bounds_right[0] + shift, 
                          T->stress_bounds_right[1] + shift, 
                          T->stress_bounds_ydir[0]  + shift, 
                          T->stress_bounds_ydir[1]  + shift);
        CUCHK(hipGetLastError());
        }
}
