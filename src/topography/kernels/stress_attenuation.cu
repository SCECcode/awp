#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <awp/definitions.h>
#include <test/test.h>
#include <topography/kernels/stress_attenuation.cuh>


__constant__ _prec d_c1;
__constant__ _prec d_c2;
__constant__ _prec d_dth;
__constant__ _prec d_dt1;
__constant__ _prec d_dh1;
__constant__ _prec d_DT;
__constant__ _prec d_DH;
__constant__ int   d_nxt;
__constant__ int   d_nyt;
__constant__ int   d_nzt;
__constant__ int   d_slice_1;
__constant__ int   d_slice_2;
__constant__ int   d_yline_1;
__constant__ int   d_yline_2;

void set_constants(const _prec dh, const _prec dt, const int nxt, const int
                nyt, const int nzt)
{
    _prec h_c1, h_c2, h_dth, h_dt1, h_dh1;

    h_c1  = 9.0/8.0;
    h_c2  = -1.0/24.0;
    h_dt1 = 1.0/dt;

    h_dth = dt/dh;
    h_dh1 = 1.0/dh;
    int slice_1  = (nyt+4+ngsl2)*(nzt+2*align);
    int slice_2  = (nyt+4+ngsl2)*(nzt+2*align)*2;
    int yline_1  = nzt+2*align;
    int yline_2  = (nzt+2*align)*2;


    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_c1),      &h_c1,    sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_c2),      &h_c2,    sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_dt1),     &h_dt1,   sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_DT),      &dt,      sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_dth),     &h_dth,   sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_dh1),     &h_dh1,   sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_DH),      &dh,      sizeof(_prec)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_nxt),     &nxt,     sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_nyt),     &nyt,     sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_nzt),     &nzt,     sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_slice_1), &slice_1, sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_slice_2), &slice_2, sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_yline_1), &yline_1, sizeof(int)));
    CUCHK(hipMemcpyToSymbol(HIP_SYMBOL(d_yline_2), &yline_2, sizeof(int)));
}

#define LDG(x) x
__launch_bounds__(512,2)
__global__ void 
dtopo_str_111(_prec*  __restrict__ xx, _prec*  __restrict__ yy, _prec*  __restrict__ zz,
           _prec*  __restrict__ xy, _prec*  __restrict__ xz, _prec*  __restrict__ yz,
       _prec*  __restrict__ r1, _prec*  __restrict__ r2,  _prec*  __restrict__ r3, 
       _prec*  __restrict__ r4, _prec*  __restrict__ r5,  _prec*  __restrict__ r6,
       _prec*  __restrict__ u1, 
       _prec*  __restrict__ v1,    
       _prec*  __restrict__ w1,    
       const float *__restrict__ f,
       const float *__restrict__ f1_1, const float *__restrict__ f1_2,
       const float *__restrict__ f1_c, const float *__restrict__ f2_1,
       const float *__restrict__ f2_2, const float *__restrict__ f2_c,
       const float *__restrict__ f_1, const float *__restrict__ f_2,
       const float *__restrict__ f_c, const float *__restrict__ g,
       const float *__restrict__ g3, const float *__restrict__ g3_c,
       const float *__restrict__ g_c,
       const _prec *__restrict__  lam,   
       const _prec *__restrict__  mu,     
       const _prec *__restrict__  qp,
       const _prec *__restrict__  coeff, 
       const _prec *__restrict__  qs, 
       const _prec *__restrict__  dcrjx, 
       const _prec *__restrict__  dcrjy, 
       const _prec *__restrict__  dcrjz, 
       const _prec *__restrict__ d_vx1, 
       const _prec *__restrict__ d_vx2, 
       const int *__restrict__ d_ww, 
       const _prec *__restrict__ d_wwo,
       int NX, int rankx, int ranky, 
       int nzt, int s_i, int e_i, int s_j, int e_j) 
{ 
  register int   i,  j,  k;
  register int   pos,     pos_ip1, pos_im2, pos_im1;
  register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
  register int   pos_jm2, pos_jm1, pos_jp1, pos_jp2;
  register int   pos_ik1, pos_jk1, pos_ijk, pos_ijk1,f_ww;
  register _prec vs1, vs2, vs3, a1, tmp, vx1,f_wwo;
  register _prec xl,  xm,  xmu1, xmu2, xmu3;
  register _prec qpa, h,   h1,   h2,   h3;
  register _prec qpaw,hw,h1w,h2w,h3w; 
  register _prec f_vx1, f_vx2,  f_dcrj, f_r,  f_dcrjy, f_dcrjz;
  register _prec f_rtmp;
  register _prec f_u1, u1_ip1, u1_ip2, u1_im1;
  register _prec f_v1, v1_im1, v1_ip1, v1_im2;
  register _prec f_w1, w1_im1, w1_im2, w1_ip1;
  _prec f_xx, f_yy, f_zz, f_xy, f_xz, f_yz;
  int maxk, mink=align+3;
    
  k    = blockIdx.x*blockDim.x+threadIdx.x+align;
  j    = blockIdx.y*blockDim.y+threadIdx.y+s_j;

  maxk = nzt + align -1;

  if (k < mink || k > maxk || j > e_j) return;
  

  i    = e_i;
  pos  = i*d_slice_1+j*d_yline_1+k;



  u1_ip1 = u1[pos+d_slice_2];
  f_u1   = u1[pos+d_slice_1];
  u1_im1 = u1[pos];    
  f_v1   = v1[pos+d_slice_1];
  v1_im1 = v1[pos];
  v1_im2 = v1[pos-d_slice_1];
  f_w1   = w1[pos+d_slice_1];
  w1_im1 = w1[pos];
  w1_im2 = w1[pos-d_slice_1];
  f_dcrjz = dcrjz[k];
  f_dcrjy = dcrjy[j];

  for(i=e_i;i>=s_i;i--)
  {
    f_vx1 = d_vx1[pos];
    f_vx2 = d_vx2[pos];
    f_ww  = d_ww[pos];
    f_wwo = d_wwo[pos];
    
    f_dcrj   = dcrjx[i]*f_dcrjy*f_dcrjz;

    pos_km2  = pos-2;
    pos_km1  = pos-1;
    pos_kp1  = pos+1;
    pos_kp2  = pos+2;
    pos_jm2  = pos-d_yline_2;
    pos_jm1  = pos-d_yline_1;
    pos_jp1  = pos+d_yline_1;
    pos_jp2  = pos+d_yline_2;
    pos_im2  = pos-d_slice_2;
    pos_im1  = pos-d_slice_1;
    pos_ip1  = pos+d_slice_1;
    pos_jk1  = pos-d_yline_1-1;
    pos_ik1  = pos+d_slice_1-1;
    pos_ijk  = pos+d_slice_1-d_yline_1;
    pos_ijk1 = pos+d_slice_1-d_yline_1-1;

    xl       = 8.0f/(  LDG(lam[pos])      + LDG(lam[pos_ip1]) + LDG(lam[pos_jm1]) + LDG(lam[pos_ijk])
                       + LDG(lam[pos_km1])  + LDG(lam[pos_ik1]) + LDG(lam[pos_jk1]) + LDG(lam[pos_ijk1]) );
    xm       = 16.0f/( LDG(mu[pos])       + LDG(mu[pos_ip1])  + LDG(mu[pos_jm1])  + LDG(mu[pos_ijk])
                       + LDG(mu[pos_km1])   + LDG(mu[pos_ik1])  + LDG(mu[pos_jk1])  + LDG(mu[pos_ijk1]) );
    xmu1     = 2.0f/(  LDG(mu[pos])       + LDG(mu[pos_km1]) );
    xmu2     = 2.0/(  LDG(mu[pos])       + LDG(mu[pos_jm1]) );
    xmu3     = 2.0/(  LDG(mu[pos])       + LDG(mu[pos_ip1]) );
    xl       = xl  +  xm;
    qpa      = 0.0625f*( LDG(qp[pos])     + LDG(qp[pos_ip1]) + LDG(qp[pos_jm1]) + LDG(qp[pos_ijk])
                         + LDG(qp[pos_km1]) + LDG(qp[pos_ik1]) + LDG(qp[pos_jk1]) + LDG(qp[pos_ijk1]) );

    if(1.0f/(qpa*2.0f)<=200.0f)
    {
      qpaw=coeff[f_ww*2-2]*(2.*qpa)*(2.*qpa)+coeff[f_ww*2-1]*(2.*qpa);
    }
    else {
      qpaw  = f_wwo*qpa;
    }
    qpaw=qpaw/f_wwo;


    h        = 0.0625f*( LDG(qs[pos])     + LDG(qs[pos_ip1]) + LDG(qs[pos_jm1]) + LDG(qs[pos_ijk])
                         + LDG(qs[pos_km1]) + LDG(qs[pos_ik1]) + LDG(qs[pos_jk1]) + LDG(qs[pos_ijk1]) );

    if(1.0f/(h*2.0f)<=200.0f)
    {
      hw=coeff[f_ww*2-2]*(2.0f*h)*(2.0f*h)+coeff[f_ww*2-1]*(2.0f*h);
    }
    else {
      hw  = f_wwo*h;
    }
    hw=hw/f_wwo;


    h1       = 0.250f*(  qs[pos]     + qs[pos_km1] );

    if(1.0f/(h1*2.0f)<=200.0f)
    {
      h1w=coeff[f_ww*2-2]*(2.0f*h1)*(2.0f*h1)+coeff[f_ww*2-1]*(2.0f*h1);
    }
    else {
      h1w  = f_wwo*h1;
    }
    h1w=h1w/f_wwo;

    h2       = 0.250f*(  qs[pos]     + qs[pos_jm1] );
    if(1.0f/(h2*2.0f)<=200.0f)
    {
      h2w=coeff[f_ww*2-2]*(2.0f*h2)*(2.0f*h2)+coeff[f_ww*2-1]*(2.0f*h2);
    }
    else {
      h2w  = f_wwo*h2;
    }
    h2w=h2w/f_wwo;


    h3       = 0.250f*(  qs[pos]     + qs[pos_ip1] );
    if(1.0f/(h3*2.0f)<=200.0f)
    {
      h3w=coeff[f_ww*2-2]*(2.0f*h3)*(2.0f*h3)+coeff[f_ww*2-1]*(2.0f*h3);
    }
    else {
      h3w  = f_wwo*h3;
    }
    h3w=h3w/f_wwo;

    h        = -xm*hw*d_dh1;
    h1       = -xmu1*h1w*d_dh1;
    h2       = -xmu2*h2w*d_dh1;
    h3       = -xmu3*h3w*d_dh1;


    qpa      = -qpaw*xl*d_dh1;

    xm       = xm*d_dth;
    xmu1     = xmu1*d_dth;
    xmu2     = xmu2*d_dth;
    xmu3     = xmu3*d_dth;
    xl       = xl*d_dth;
    h        = h*f_vx1;
    h1       = h1*f_vx1;
    h2       = h2*f_vx1;
    h3       = h3*f_vx1;
    qpa      = qpa*f_vx1;

    xm       = xm+d_DT*h;
    xmu1     = xmu1+d_DT*h1;
    xmu2     = xmu2+d_DT*h2;
    xmu3     = xmu3+d_DT*h3;
    vx1      = d_DT*(1+f_vx2*f_vx1);
        
    u1_ip2   = u1_ip1;
    u1_ip1   = f_u1;
    f_u1     = u1_im1;
    u1_im1   = u1[pos_im1];
    v1_ip1   = f_v1;
    f_v1     = v1_im1;
    v1_im1   = v1_im2;
    v1_im2   = v1[pos_im2];
    w1_ip1   = f_w1;
    f_w1     = w1_im1;
    w1_im1   = w1_im2;
    w1_im2   = w1[pos_im2];


    vs1      = d_c1*(u1_ip1 - f_u1)        + d_c2*(u1_ip2      - u1_im1);
    vs2      = d_c1*(f_v1   - v1[pos_jm1]) + d_c2*(v1[pos_jp1] - v1[pos_jm2]);
    vs3      = d_c1*(f_w1   - w1[pos_km1]) + d_c2*(w1[pos_kp1] - w1[pos_km2]);

    tmp      = xl*(vs1+vs2+vs3);

    a1       = qpa*(vs1+vs2+vs3);
    tmp      = tmp+d_DT*a1;

    f_r      = r1[pos];
    f_rtmp   = -h*(vs2+vs3) + a1; 
    f_xx     = xx[pos]  + tmp - xm*(vs2+vs3) + vx1*f_r;  
    r1[pos]  = f_vx2*f_r + f_wwo*f_rtmp;
    f_rtmp   = f_rtmp*(f_wwo-1) + f_vx2*f_r*(1-f_vx1); 
    xx[pos]  = (f_xx + d_DT*f_rtmp)*f_dcrj;

    f_r      = r2[pos];
    f_rtmp   = -h*(vs1+vs3) + a1;  
    f_yy     = (yy[pos]  + tmp - xm*(vs1+vs3) + vx1*f_r)*f_dcrj;
    r2[pos]  = f_vx2*f_r + f_wwo*f_rtmp; 
    f_rtmp   = f_rtmp*(f_wwo-1) + f_vx2*f_r*(1-f_vx1); 
    yy[pos]  = (f_yy + d_DT*f_rtmp)*f_dcrj;
	
    f_r      = r3[pos];
    f_rtmp   = -h*(vs1+vs2) + a1;
    f_zz     = (zz[pos]  + tmp - xm*(vs1+vs2) + vx1*f_r)*f_dcrj;
    r3[pos]  = f_vx2*f_r + f_wwo*f_rtmp;
    f_rtmp   = f_rtmp*(f_wwo-1.0f) + f_vx2*f_r*(1.0f-f_vx1);  
    zz[pos]  = (f_zz + d_DT*f_rtmp)*f_dcrj;


    vs1      = d_c1*(u1[pos_jp1] - f_u1)   + d_c2*(u1[pos_jp2] - u1[pos_jm1]);
    vs2      = d_c1*(f_v1        - v1_im1) + d_c2*(v1_ip1      - v1_im2);
    f_r      = r4[pos];
    f_rtmp   = h1*(vs1+vs2); 
    f_xy     = xy[pos]  + xmu1*(vs1+vs2) + vx1*f_r;
    r4[pos]  = f_vx2*f_r + f_wwo*f_rtmp; 
    f_rtmp   = f_rtmp*(f_wwo-1) + f_vx2*f_r*(1-f_vx1);
    xy[pos]  = (f_xy + d_DT*f_rtmp)*f_dcrj;


    vs1     = d_c1*(u1[pos_kp1] - f_u1)   + d_c2*(u1[pos_kp2] - u1[pos_km1]);
    vs2     = d_c1*(f_w1        - w1_im1) + d_c2*(w1_ip1      - w1_im2);
    f_r     = r5[pos];
    f_rtmp  = h2*(vs1+vs2);
    f_xz    = xz[pos]  + xmu2*(vs1+vs2) + vx1*f_r; 
    r5[pos] = f_vx2*f_r + f_wwo*f_rtmp; 
    f_rtmp  = f_rtmp*(f_wwo-1.0f) + f_vx2*f_r*(1.0f-f_vx1); 
    xz[pos] = (f_xz + d_DT*f_rtmp)*f_dcrj;
	 
    vs1     = d_c1*(v1[pos_kp1] - f_v1) + d_c2*(v1[pos_kp2] - v1[pos_km1]);
    vs2     = d_c1*(w1[pos_jp1] - f_w1) + d_c2*(w1[pos_jp2] - w1[pos_jm1]);
    f_r     = r6[pos];
    f_rtmp  = h3*(vs1+vs2);
    f_yz    = yz[pos]  + xmu3*(vs1+vs2) + vx1*f_r;
    r6[pos] = f_vx2*f_r + f_wwo*f_rtmp;
    f_rtmp  = f_rtmp*(f_wwo-1.0f) + f_vx2*f_r*(1.0f-f_vx1); 
    yz[pos] = (f_yz + d_DT*f_rtmp)*f_dcrj; 

  if (k == mink && i == s_i) {
        printf("xx = %g vs1 = %g  \n", xx[pos], vx1);
  }

    pos     = pos_im1;
  }
}

