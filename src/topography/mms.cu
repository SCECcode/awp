#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <topography/mms.cuh>
#include <awp/pmcl3d_cons.h>

// Background values (P-wave speed, S-wave speed, density)
static float scp0, scs0, srho0;
// Perturbation values (P-wave speed, S-wave speed, density)
static float sdcp, sdcs, sdrho;
// Wave mode
static int smode;

__global__ void material_properties(
              const int nxt, const int nyt,
              const int nzt, float *d_d1, float *d_lam,
              float *d_mu, float *d_qp, float *d_qs,
              const float lam0, const float mu0, const float rho0, 
              const float dlam, const float dmu, const float drho, 
              const float mode, const float h, const int px, const int py
              ) {

                int i = threadIdx.z + blockDim.z * blockIdx.z;
                int j = threadIdx.y + blockDim.y * blockIdx.y;
                int k = align + threadIdx.x + blockDim.x * blockIdx.x;

                if (i < ngsl / 2 + 2 || i >= nxt + ngsl + 2) return;
                if (j < ngsl / 2 + 2 || j >= nyt + ngsl + 2) return;
                if (k >= align + nzt) return;
                
                float Lx =  (nxt - 1) * h;
                float Ly =  (nyt - 1) * h;
                float Lz =  (nzt - 1) * h;

                float x = (i - ngsl - 2) * h + px * Lx;
                float y = (j - ngsl - 2) * h + py * Ly;
                float z = h * k;

                int line = 2 * align + nzt;
                int slice = line * (4 + 2 * ngsl + nyt);
                int pos = k + line * j + slice * i;

                float S = sin(M_PI * mode * x / Lx) *
                          sin(M_PI * mode * y / Ly) * sin(M_PI * mode * z / Lz);

                //FIXME
                S = 0.0f;
                d_d1[pos] = rho0 + drho * S;
                d_lam[pos] = 1.0f / (lam0 + dlam * S);
                d_mu[pos] = 1.0f /  (mu0 + dmu * S);
                d_qp[pos] = 1e-10;
                d_qs[pos] = 1e-10;
                if (i == 20 && j == 20 && k == 32) printf("rho = %g lam = %g mu = %g \n", d_d1[pos], d_lam[pos], d_mu[pos]);
                                        
                                        
}

void mms_init(const char *MMSFILE,
        const int *nxt, const int *nyt,
              const int *nzt, const int ngrids, float **d_d1, float **d_lam,
              float **d_mu,
              float **d_qp, float **d_qs,
              float **d_vx, float **d_vy, float **d_vz,
              float **d_xx, float **d_yy, float **d_zz, float **d_xy,
              float **d_xz, float **d_yz, int px, int py, const float *h)
{


        FILE *fh = fopen(MMSFILE, "r");
        if (!fh)  {
         if (px == 0 && py == 0) {
                 fprintf(stderr, "Failed to open: %s \n", MMSFILE);
                 exit(-1);
        }
                return; 
        }

 

        int parsed = fscanf(fh, "%f %f %f %f %f %f %d\n", &scp0, &scs0, &srho0, &sdcp, &sdcs, &sdrho, &smode);
        if (parsed != 7 && px == 0 && py == 0)
                 fprintf(stderr, "Failed to parse: %s \n", MMSFILE);

        if (px == 0 && py == 0) {
                printf("Done reading mms input file\n");
                printf("Settings: \n");
                printf("        cp0 = %g cs0 = %g rho0 = %g \n", scp0, scs0, srho0);
                printf("        dcp = %g dcs = %g drho = %g \n", sdcp, sdcs, sdrho);
                printf("        mode = %d \n", smode);


        }


        dim3 threads (32, 4, 1);
        for (int p = 0; p < ngrids; ++p) {
        
                int mz = nzt[p];
                int my = nyt[p] + 2 * ngsl + 4;
                int mx = nxt[p] + 2 * ngsl + 4;

                float mu0 = scs0 * scs0 * srho0;
                float dmu = sdcs * sdcs * sdrho;
                float lam0 = scp0 * scp0 * srho0 - 2.0 * scs0 * scs0 * srho0;
                float dlam = sdcp * sdcp * sdrho - 2.0 * sdcs * sdcs * sdrho;
                printf("mu0 = %g lam0 = %g dlam = %g dmu = %g \n", mu0, lam0, dlam, dmu);

                if (px == 0 && py == 0) printf("Setting material properties for grid = %d \n", p);
                // Set material properties
                dim3 blocks( (mz - 1) / threads.x + 1, (my - 1) / threads.y + 1, (mx - 1) / threads.z + 1);
                material_properties<<<blocks, threads>>>(nxt[p], nyt[p], nzt[p], d_d1[p], d_lam[p], d_mu[p], d_qp[p], d_qs[p],
                                lam0, mu0, srho0, dlam, dmu, sdrho, smode, h[p], px, py);

                hipError_t cerr;
                CUCHK(cerr=hipGetLastError());



        }

                if (px == 0 && py == 0) printf("MMS initialization done. \n");
}

